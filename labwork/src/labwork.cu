#include "hip/hip_runtime.h"
#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv) {
    printf("USTH ICT Master 2018, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum) {
        case 1:
			timer.start();
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
            labwork.labwork1_OpenMP();
            labwork.saveOutputImage("labwork2-openmp-out.jpg");
			printf("labwork 1 GPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
            break;
        case 4:
            labwork.labwork4_GPU();
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
            break;
        case 5:
            labwork.labwork5_CPU();
            labwork.saveOutputImage("labwork5-cpu-out.jpg");
            labwork.labwork5_GPU();
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
    printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {		// let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP() {
int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
	//#pragma omp parallel for schedule(static,1) num_threads(8)
	//#pragma omp parallel for schedule(dynamic,1) num_threads(8)
	#pragma omp target teams num_teams(4)
    for (int j = 0; j < 100; j++) {		// let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {
    int numberOfDevices = 0;
	hipGetDeviceCount(&numberOfDevices);
  printf("Number of devices: %d\n", numberOfDevices); 
  for(int i=0; i<numberOfDevices; i++) { 
      hipDeviceProp_t prop; 
      hipGetDeviceProperties(&prop, i); 
      printf("Device %d\n", i); 
      printf("Name: %s\n", prop.name);
	  printf("Cores %d\n",getSPcores(prop)); 
      printf("Clockrate: %d\n", prop.clockRate);
	  }
}


__global__ void rgb2grayCUDA(uchar3 *input, uchar3 *output) {
int tid = threadIdx.x + blockIdx.x * blockDim.x;
output[tid].x = (input[tid].x + input[tid].y +
input[tid].z) / 3;
output[tid].z = output[tid].y = output[tid].x;
}

void Labwork::labwork3_GPU() {
//  number of pixels 
int pixelCount = inputImage->width * inputImage->height;
uchar3 *devInput;
uchar3 *devGray;
int regionSize = 1024;
int numBlock = pixelCount / regionSize;

 hipMalloc(&devInput, pixelCount * sizeof(uchar3));
 hipMalloc(&devGray, pixelCount * sizeof(float));
 //copy from host to device
 hipMemcpy(devInput,inputImage->buffer,pixelCount * sizeof(uchar3),hipMemcpyHostToDevice);
 //launch the kernel
 rgb2grayCUDA<<<numBlock, regionSize>>>(devInput, devGray);
 outputImage = (char*) malloc(pixelCount * sizeof(char) * 3);
 hipMemcpy(outputImage, devGray,pixelCount * sizeof(float),hipMemcpyDeviceToHost);
 //free memory
 hipFree(devInput);
 hipFree(devGray);
}
__global__ void rgb2grayCUDA2D(uchar3 *input, uchar3 *output,int width, int height) {
int tidX = threadIdx.x + blockIdx.x * blockDim.x;
if(tidX >= width) return;
int  tidY = threadIdx.y + blockIdx.y * blockDim.y;
if(tidY >= height) return;
int tid =  (tidX + tidY) * width;
output[tid].x = (input[tid].x + input[tid].y +input[tid].z) / 3;
output[tid].z = output[tid].y = output[tid].x;
}

void Labwork::labwork4_GPU() {
//  number of pixels 
int pixelCount = inputImage->width * inputImage->height;
uchar3 *devInput;
uchar3 *devGray;
dim3 regionSize = dim3((inputImage->width + 7)/8, (inputImage->height + 7)/8);
dim3 numBlock = dim3(8, 8);
//grayscale<<<gridSize, blockSize>>>(devInput, devOutput);

 hipMalloc(&devInput, pixelCount * sizeof(uchar3));
 hipMalloc(&devGray, pixelCount * sizeof(uchar3));
 //copy from host to device
 hipMemcpy(devInput,inputImage->buffer,pixelCount * sizeof(uchar3),hipMemcpyHostToDevice);
 //launch the kernel
 rgb2grayCUDA2D<<<numBlock, regionSize>>>(devInput, devGray,inputImage->width,inputImage->height);
 outputImage = (char*) malloc(pixelCount * sizeof(char) * 3);
 hipMemcpy(outputImage, devGray,pixelCount * sizeof(uchar3),hipMemcpyDeviceToHost);
 //free memory
 hipFree(devInput);
 hipFree(devGray);
}

// CPU implementation of Gaussian Blur
void Labwork::labwork5_CPU() {
    int kernel[] = { 0, 0, 1, 2, 1, 0, 0,  
                     0, 3, 13, 22, 13, 3, 0,  
                     1, 13, 59, 97, 59, 13, 1,  
                     2, 22, 97, 159, 97, 22, 2,  
                     1, 13, 59, 97, 59, 13, 1,  
                     0, 3, 13, 22, 13, 3, 0,
                     0, 0, 1, 2, 1, 0, 0 };
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = (char*) malloc(pixelCount * sizeof(char) * 3);
    for (int row = 0; row < inputImage->height; row++) {
        for (int col = 0; col < inputImage->width; col++) {
            int sum = 0;
            int c = 0;
            for (int y = -3; y <= 3; y++) {
                for (int x = -3; x <= 3; x++) {
                    int i = col + x;
                    int j = row + y;
                    if (i < 0) continue;
                    if (i >= inputImage->width) continue;
                    if (j < 0) continue;
                    if (j >= inputImage->height) continue;
                    int tid = j * inputImage->width + i;
                    unsigned char gray = (inputImage->buffer[tid * 3] + inputImage->buffer[tid * 3 + 1] + inputImage->buffer[tid * 3 + 2])/3;
                    int coefficient = kernel[(y+3) * 7 + x + 3];
                    sum = sum + gray * coefficient;
                    c += coefficient;
                }
            }
            sum /= c;
            int posOut = row * inputImage->width + col;
            outputImage[posOut * 3] = outputImage[posOut * 3 + 1] = outputImage[posOut * 3 + 2] = sum;
        }
    }
}

void Labwork::labwork5_GPU() {
    
}

void Labwork::labwork6_GPU() {

}

void Labwork::labwork7_GPU() {

}

void Labwork::labwork8_GPU() {

}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU() {

}
